
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <algorithm>
#include <functional>
#include <vector>
#include <numeric>
#include <chrono>
#include <iostream>

#define N 1024

// gpu kernel callable from device and host
__global__ 
void mult_gpu(int* d_a_in, int* d_b_in, int* d_c_out, int NN)
{
    // compute the 1d thread id based on the block and thread ids
    int row= blockIdx.x * blockDim.x + threadIdx.x;
    int col= blockIdx.y * blockDim.y + threadIdx.y;

    int tmp = 0 ; 
    // do a bound check and return if out of bounds
    for( int k = 0 ; k < NN ; k++ )
    {
        tmp += d_a_in[row*NN + k] * d_b_in[k*NN + col];
    }

    // add the two input arrays and store the result in the output array
     d_c_out[row*NN + col] = tmp; 
};

void mult_cpu(int* h_a_in, int* h_b_in, int* h_c_out, int NN)
{
    for(int i=0; i<NN; i++)
    {
        for(int j=0; j<NN; j++)
        {
            for(int k=0; k<NN; k++)
            {
                h_c_out[i*NN+j] += h_a_in[i*NN+k] * h_b_in[k*NN+j];
            }
        }
    }
};

int main()
{
    // create device pointers 
    std::vector<int> h_a_in(N*N);
    std::vector<int> h_b_in(N*N); 
    std::vector<int> h_c_out(N*N); 
    std::vector<int> h_c_out_gpu(N*N); 

    // iniatilize the matrices
    std::iota(h_a_in.begin(), h_a_in.end(), 0);
    std::iota(h_b_in.begin(), h_b_in.end(), 0);
	
    // call the CPU routine
    auto start_cpu = std::chrono::high_resolution_clock::now();
	mult_cpu(h_a_in.data(), h_b_in.data(), h_c_out.data(), N); 
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff_cpu = end_cpu - start_cpu;
    std::cout << "CPU time: " << diff_cpu.count() << " s\n";


    //allocate device memory
    int *d_a_in, *d_b_in, *d_c_out;  
    hipMalloc(&d_a_in, N*N*sizeof(int));
    hipMalloc(&d_b_in, N*N*sizeof(int));
    hipMalloc(&d_c_out, N*N*sizeof(int));

    //copy the pointer to the device
    hipMemcpy(d_a_in, h_a_in.data(), N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_in, h_b_in.data(),N*N*sizeof(int), hipMemcpyHostToDevice);
    
    // assume N is divisible by 32
    static_assert( N % 32 == 0, "N must be divisible by 32");
    dim3 block(32,32,1);
    dim3 grid(N/32,N/32,1);

    auto start_gpu = std::chrono::high_resolution_clock::now();
    // call the gpu kernel 
    mult_gpu<<<grid,block>>>(d_a_in, d_b_in, d_c_out, N);
    // wait for gpu to finish
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff_gpu = end_gpu - start_gpu;
    std::cout << "GPU time: " << diff_gpu.count() << " s\n";

    hipMemcpy(h_c_out_gpu.data(), d_c_out, N*N*sizeof(int), hipMemcpyDeviceToHost);

    // check they are equal 
    if( std::equal(h_c_out.data(), h_c_out.data()+N*N, h_c_out_gpu.data()) ) printf("Success\n");
    else printf("Error\n");

    // Check for CUDA errors
    hipError_t cudaErr = hipGetLastError();
    if(cudaErr != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(cudaErr));
    }

    // free memeory of the device and host
    hipFree(d_a_in);
    hipFree(d_b_in);
    hipFree(d_c_out);


    return 0;
}