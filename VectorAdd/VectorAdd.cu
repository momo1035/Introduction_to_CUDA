
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <algorithm>
#include <functional>

#define N 1000

// gpu kernel callable from device and host
__global__ 
void add_gpu(int* d_a_in, int* d_b_in, int* d_c_out, int NN)
{
    // compute the 1d thread id based on the block and thread ids
    int tid= blockIdx.x * blockDim.x + threadIdx.x;

    // do a bound check and return if out of bounds
    if ( tid < NN )
    {
        // add the two input arrays and store the result in the output array
        d_c_out[tid] = d_a_in[tid] + d_b_in[tid];
    }
};

int main()
{
    // create device pointers 
    int* h_a_in = (int*)malloc(N*sizeof(int));
    int *h_b_in = (int*)malloc(N*sizeof(int));
    int *h_c_out = (int*)malloc(N*sizeof(int));
	int* h_c_out_gpu = (int*)malloc(N*sizeof(int));


    // fill the input arrays with some data
    for(int i=0; i<N; i++)
    {
        h_a_in[i] = i;
        h_b_in[i] = i+1;
    }

    //allocate device memory
    int *d_a_in, *d_b_in, *d_c_out; 
    hipMalloc(&d_a_in, N*sizeof(int));
    hipMalloc(&d_b_in, N*sizeof(int));
    hipMalloc(&d_c_out, N*sizeof(int));

    //copy the pointer to the device
    hipMemcpy(d_a_in, h_a_in, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_in, h_b_in, N*sizeof(int), hipMemcpyHostToDevice);

    //compute the sum with std libray
    std::transform(h_a_in, h_a_in+N, h_b_in, h_c_out, std::plus<int>());

    // call the gpu kernel 
    add_gpu<<<1,N>>>(d_a_in, d_b_in, d_c_out, N);
    // wait for gpu to finish
    hipDeviceSynchronize();
	
	// copy the memeory back to cpu to compare
	hipMemcpy(h_c_out_gpu, d_c_out, N*sizeof(int), hipMemcpyDeviceToHost);
		
    // check they are equal 
    if( std::equal(h_c_out, h_c_out+N, h_c_out_gpu) ) printf("Success\n");
    else printf("Error\n");

    // Check for CUDA errors
    hipError_t cudaErr = hipGetLastError();
    if(cudaErr != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(cudaErr));
    }

    // free memeory of the device and host
    free(h_a_in);
    free(h_b_in);
    free(h_c_out);
	free(h_c_out_gpu);
    hipFree(d_a_in);
    hipFree(d_b_in);
    hipFree(d_c_out);


    return 0;
}