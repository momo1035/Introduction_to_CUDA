
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
// gpu kernel callable from device and host
__global__ 
void gpuhelloword(void)
{
    printf("Hello World from GPU\n");
};
int main()
{
    // call gpu kernel
    gpuhelloword<<<1,16>>>();
    // wait for gpu to finish
    hipDeviceSynchronize();
    // Check for CUDA errors
    hipError_t cudaErr = hipGetLastError();
    if(cudaErr != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(cudaErr));
    }
    return 0;
}